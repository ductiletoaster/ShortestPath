#include "hip/hip_runtime.h"
/**
 * Author: Brian Gebel
 * Version: 12.10.2011
 * Description:
 */

/**   
 *	For testing parallelization speeds with different memory types.
 *	The new version may prove to be faster, will test in future version.
 *
 *	OLD VERSION(USING GLOBAL DEVICE MEM)
 *	int *hostMatrix = (int*)malloc(sizeof(int)*SIZE)	
 * 	int *devMatrix;
 *	hipMalloc((void**) &devMatrix,sizeof(int)*SIZE);
 * 	hipMemcpy(devMatrix, hostMatrix, sizeof(int)*SIZE, hipMemcpyHostToDevice);
 *
 *	NEW VERSION(USING CONSTANT DEVICE MEM)
 *	_constant_ int devMatrix[SIZE];
 *	hipMemcpyToSymbol(HIP_SYMBOL(devMatrix),hostMatrix,sizeof(int)*SIZE);
 *
 * Seems their is an issue when we have more than 2 blocks.
 * Threads may be run over the top of each other should debug
 * Also when determining the number of blocks we could easily optimise that
 */

#include <iostream>
#include <stdio.h>
#include <windows.h>
#include "hip/hip_runtime.h"
#include ""
#include "parallelpath.cuh"

using namespace std;
using std::cout;
using std::endl;

__global__ void findBestPathKernel(int *deviceMatrix, int *deviceBestToDiagPathSum, int *deviceBestFromDiagPathSum, unsigned int *deviceBestToDiagPathCode, unsigned int *deviceBestFromDiagPathCode, int N) {

	// Iterate through path codes, updating bestToDiag and bestFromDiag arrays
	int toDiagPathSum, toDiagRow, toDiagCol;
	int fromDiagPathSum, fromDiagRow, fromDiagCol;

	unsigned int maxPath = ((unsigned int) 1 << N - 1);
	unsigned int currentPathCode = blockIdx.x * threadsPerBlock + threadIdx.x;
	unsigned int nextPathCode = blockDim.x * gridDim.x;

	while (currentPathCode <= maxPath) {

		unsigned int ui = currentPathCode;

		toDiagPathSum = deviceMatrix[0];
		toDiagRow = 0;
		toDiagCol = 0;

		fromDiagPathSum = deviceMatrix[(N - 1) * N + (N - 1)];
		fromDiagRow = N - 1;
		fromDiagCol = N - 1;

		for ( int i = 0 ; i < N - 1 ; i++ ) {
			if ( ui % 2 == 0) {
				toDiagCol++; // Horizontal move
				fromDiagCol--;
			} else {
				toDiagRow++;
				fromDiagRow-- ;	// Vertical move
			}

			toDiagPathSum += deviceMatrix[(toDiagRow * N) + toDiagCol];
			fromDiagPathSum += deviceMatrix[(fromDiagRow * N) + fromDiagCol];
			ui = ui >> 1;
		}
		
		if (toDiagPathSum < deviceBestToDiagPathSum[toDiagRow] ) {
			deviceBestToDiagPathSum[toDiagRow] = toDiagPathSum;
			deviceBestToDiagPathCode[toDiagRow] = currentPathCode;
		}

		if (fromDiagPathSum < deviceBestFromDiagPathSum[fromDiagRow]) {
			deviceBestFromDiagPathSum[fromDiagRow] = fromDiagPathSum;
			deviceBestFromDiagPathCode[fromDiagRow] = currentPathCode;
		}

		// Next Path Code
		currentPathCode += nextPathCode;
	}
}

void findBestPathParallel(int N) {
	// Memory Allocations
	unsigned int matrixMemSize = sizeof(int) * N * N;
	unsigned int arrayMemSize = sizeof(int) * N;
	unsigned int unsignedArrayMemSize = sizeof(unsigned int) * N;

	// Allocate Memory on the host
	int *hostMatrix;
    hostMatrix = (int*) malloc(sizeof(int) * N * N);

	// Create and initialize matrix
	for (int i = 0 ; i < N * N ; i++)
		hostMatrix[i] = 99;

	// Create path along edges of matrix
	if (N == 16) {
		setMatrixParallelPath(hostMatrix);
	} else {
		for (int i = 0 ; i < N; i++) {
			hostMatrix[i] = 1;
			hostMatrix[i * N + (N - 1)] = 1;
		}
	}

	// Create and initialize bestTodiag and bestfromDiag arrays on host
	int *hostBestToDiagPathSum;
	hostBestToDiagPathSum = (int*) malloc(sizeof(int) * N);

	int *hostBestFromDiagPathSum;
	hostBestFromDiagPathSum = (int*) malloc(sizeof(int) * N);

	unsigned int *hostBestToDiagPathCode; //used unsigned int to avoid 2's complent
	hostBestToDiagPathCode = (unsigned int*) malloc(sizeof(unsigned int) * N);

	unsigned int *hostBestFromDiagPathCode; //used unsigned int to avoid 2's complent
	hostBestFromDiagPathCode = (unsigned int*) malloc(sizeof(unsigned int) * N);

	int biggerThanMaxPathSum = 256 * N + 1;
	for ( int i = 0 ; i < N ; i++ ) {
		hostBestToDiagPathSum[i]  = biggerThanMaxPathSum ;
		hostBestFromDiagPathSum[i] = biggerThanMaxPathSum ;
	}

	// Allocate Memory on the device for matrices
	int *deviceMatrix;
    hipMalloc((void**) &deviceMatrix, matrixMemSize);
	hipMemcpy(deviceMatrix, hostMatrix, matrixMemSize, hipMemcpyHostToDevice);

	// Allocate Memory on the device for arrays
	int *deviceBestToDiagPathSum;
    hipMalloc((void**) &deviceBestToDiagPathSum, arrayMemSize);
	hipMemcpy(deviceBestToDiagPathSum, hostBestToDiagPathSum, arrayMemSize, hipMemcpyHostToDevice);

	int *deviceBestFromDiagPathSum;
	hipMalloc((void**) &deviceBestFromDiagPathSum, arrayMemSize);
	hipMemcpy(deviceBestFromDiagPathSum, hostBestFromDiagPathSum, arrayMemSize, hipMemcpyHostToDevice);

	// Allocate Memory on the device for unsigned arrays
	unsigned int *deviceBestToDiagPathCode; //used unsigned int to avoid 2's complent
	hipMalloc((void**) &deviceBestToDiagPathCode, unsignedArrayMemSize);
	hipMemcpy(deviceBestToDiagPathCode, hostBestToDiagPathCode, unsignedArrayMemSize, hipMemcpyHostToDevice);

	unsigned int *deviceBestFromDiagPathCode; //used unsigned int to avoid 2's complent
	hipMalloc((void**) &deviceBestFromDiagPathCode, unsignedArrayMemSize);
	hipMemcpy(deviceBestFromDiagPathCode, hostBestFromDiagPathCode, unsignedArrayMemSize, hipMemcpyHostToDevice);

	// Execution Parameters
	int totalBlocks = (((unsigned int)1 << (N - 1)) + threadsPerBlock) / threadsPerBlock;
	if (totalBlocks > maxBlocks) totalBlocks = maxBlocks;

	// Launch a kernel on the GPU with one thread for each element.
	findBestPathKernel<<< totalBlocks, threadsPerBlock >>>(deviceMatrix, deviceBestToDiagPathSum, deviceBestFromDiagPathSum, deviceBestToDiagPathCode, deviceBestFromDiagPathCode, N);

	// Copy result from device to host
	hipMemcpy(hostMatrix, deviceMatrix, matrixMemSize, hipMemcpyDeviceToHost);
	hipMemcpy(hostBestToDiagPathSum, deviceBestToDiagPathSum, arrayMemSize, hipMemcpyDeviceToHost);
	hipMemcpy(hostBestFromDiagPathSum, deviceBestFromDiagPathSum, arrayMemSize, hipMemcpyDeviceToHost);
	hipMemcpy(hostBestToDiagPathCode, deviceBestToDiagPathCode, unsignedArrayMemSize, hipMemcpyDeviceToHost);
	hipMemcpy(hostBestFromDiagPathCode, deviceBestFromDiagPathCode, unsignedArrayMemSize, hipMemcpyDeviceToHost);

	// Examine bestToDiag and bestFromDiag arrays to decide best path
	int bestPathSum = biggerThanMaxPathSum;
	unsigned int bestPathCodePrefix ;
	unsigned int bestPathCodeSuffix ;

	int tempSum ;

	for (int i = 0 ; i < N ; i++) {
		tempSum = hostBestToDiagPathSum[i] + hostBestFromDiagPathSum[i] - hostMatrix[i * N + (N - 1 - i)];
		if (tempSum < bestPathSum) {
			bestPathSum = tempSum;
			bestPathCodePrefix = hostBestToDiagPathCode[i];
			bestPathCodeSuffix = hostBestFromDiagPathCode[i];
		}
	}

	hostMatrix[0] = -hostMatrix [0];
	int toDiagRow = 0;
	int toDiagCol = 0;
	unsigned ui = bestPathCodePrefix;
	
	for ( int i = 0 ; i < N - 1 ; i++ ) {
		if (ui % 2 == 0 )
			toDiagCol++ ;
		else
			toDiagRow++ ;
		if (hostMatrix[toDiagCol + (toDiagRow * N)] > 0) hostMatrix[toDiagCol + (toDiagRow * N)] = -hostMatrix[toDiagCol + (toDiagRow * N)];
		ui = ui >> 1;
	}

	hostMatrix[(N - 1) * N + (N - 1)] = -hostMatrix [(N - 1) * N + (N - 1)] ;
	int fromDiagRow = N - 1;
	int fromDiagCol = N - 1;
	ui = bestPathCodeSuffix;
	
	for ( int i = 0 ; i < N - 1 ; i++ ) {
		if (ui % 2 == 0 )
			fromDiagCol--;
		else
			fromDiagRow--;
		if (hostMatrix[fromDiagCol + (fromDiagRow * N)] > 0) hostMatrix[fromDiagCol + (fromDiagRow * N)] = -hostMatrix[fromDiagCol + (fromDiagRow * N)];
		ui = ui >> 1;
	}

	displayBestPathParallel(hostMatrix, N, bestPathSum);

	// Free up memory
	free(hostMatrix);
	free(hostBestToDiagPathSum);
	free(hostBestFromDiagPathSum);
	free(hostBestToDiagPathCode);
	free(hostBestFromDiagPathCode);
	hipFree(deviceMatrix);
	hipFree(deviceBestToDiagPathSum);
	hipFree(deviceBestFromDiagPathSum);
	hipFree(deviceBestToDiagPathCode);
	hipFree(deviceBestFromDiagPathCode);
}

void displayBestPathParallel(int *hostMatrix, int N, int bestPathSum) {
	// Allows for colors in cmd window
	CONSOLE_SCREEN_BUFFER_INFO *ConsoleInfo = new CONSOLE_SCREEN_BUFFER_INFO();
    GetConsoleScreenBufferInfo(GetStdHandle(STD_OUTPUT_HANDLE), ConsoleInfo);
    WORD OriginalColors = ConsoleInfo->wAttributes;

	// Builds Visual Matrix
	for (int row = 0 ; row < N ; row++) {
		for ( int col = 0 ; col < N ; col++)
			if (hostMatrix[row * N + col] < 0) {
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), FOREGROUND_RED);
				cout << -hostMatrix[row * N + col];
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), OriginalColors);
			} else
				cout << "@";
		cout << endl;
	}
	cout << endl <<"Best Path Sum: " << bestPathSum << endl;
}

void setMatrixParallelPath(int *Matrix) {
	Matrix[0] = 1; // Row 0
	Matrix[1] = 3;
	Matrix[2] = 3;
	Matrix[3] = 3;
	Matrix[4] = 3;

	Matrix[16] = 1;
	Matrix[20] = 3;

	Matrix[32] = 1;
	Matrix[36] = 3;
	Matrix[45] = 1;

	Matrix[48] = 1;
	Matrix[52] = 3;
	Matrix[61] = 1;
	Matrix[62] = 1; 

	Matrix[64] = 1; // Row 4
	Matrix[68] = 3;
	Matrix[69] = 3;
	Matrix[70] = 3;
	Matrix[78] = 1;
	Matrix[79] = 1;

	Matrix[80] = 1;
	Matrix[81] = 1;
	Matrix[86] = 3;
	Matrix[95] = 1;

	Matrix[97] = 1;
	Matrix[102] = 3;
	Matrix[111] = 1;

	Matrix[113] = 1;
	Matrix[118] = 3;
	Matrix[127] = 1;

	Matrix[129] = 1; // Row 8
	Matrix[134] = 3;
	Matrix[135] = 2;
	Matrix[143] = 1;

	Matrix[145] = 1;
	Matrix[151] = 2;
	Matrix[152] = 2;
	Matrix[153] = 2;
	Matrix[154] = 2;
	Matrix[159] = 1;

	Matrix[161] = 1;
	Matrix[162] = 1;
	Matrix[170] = 2;
	Matrix[175] = 1;

	Matrix[178] = 1;
	Matrix[186] = 2;
	Matrix[191] = 1;

	Matrix[194] = 1; // Row 12
	Matrix[195] = 1;
	Matrix[202] = 2;
	Matrix[207] = 1;

	Matrix[218] = 2;
	Matrix[219] = 2;
	Matrix[220] = 2;
	Matrix[224] = 1;

	Matrix[236] = 2;
	Matrix[237] = 2;
	Matrix[239] = 1;

	Matrix[253] = 2;
	Matrix[254] = 2;
	Matrix[255] = 1;
}